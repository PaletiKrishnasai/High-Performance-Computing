#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<unistd.h>
 
//#define Threads 2
#define n_size 800
 
__global__ void add(float *a, float *b, float *c) {
 
 int index=threadIdx.x+blockIdx.x*blockDim.x;
 c[index]=a[index]+b[index];
}
 
int main() {
  
float a[n_size][n_size], b[n_size][n_size],c[n_size][n_size];
hipEvent_t start, end;
 
// host copies of variables a, b & c
float *d_a, *d_b, *d_c;
 
// device copies of variables a, b & c
int size = n_size*n_size*sizeof(float);
 
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
 
// Create Event for time
hipEventCreate(&start);
hipEventCreate(&end);
 
 
// Setup input values 
int ch=0;
srand(time(NULL));
   if(ch==0)
   {
       for(int i=0;i<n_size;i++)
       {
           for(int j=0;j<n_size;j++)
           {
               a[i][j]=((float)rand()/(float)(RAND_MAX)) * 5.0;
               b[i][j]=((float)rand()/(float)(RAND_MAX)) * 5.0;
           }
          
 
       }
   }
   else
   {
       for(int i=0;i<n_size;i++)
       {
           for(int j=0;j<n_size;j++)
           {
               a[i][j]=(i+j+1);
               b[i][j]=(i+j+1);
           }
            
       }
   }
  
 
 
 
// Copy inputs to device
hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
int Thread[]={1,2,4,6,8,10,12,16,20,32,64,128,150};
int thread_arr_size=13;
 
for(int i=0;i<thread_arr_size;i++)
{       sleep(1);
       int Threads=Thread[i];
       hipEventRecord(start);
 
       // Launch add() kernel on GPU
       add<<<(n_size*n_size)/Threads,Threads>>>(d_a, d_b, d_c);
 
       hipDeviceSynchronize();
       hipEventRecord(end);
       hipEventSynchronize(end);
 
       float time = 0;
       hipEventElapsedTime(&time, start, end);
 
       // Copy result back to host
       hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
       if(err!=hipSuccess) {
           printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
       }
 
       int flag=0;
       for(int i=0;i<n_size;i++)
       { 
           for(int j=0;j<n_size;j++)
           {
               //printf("Result[%d][%d]=%f\n",i+1,j+1,c[i][j]);
               if(c[i][j]!=(a[i][j]+b[i][j]))
               { 
                   flag=1;
               break;
           }
           }
       }
       if(flag==0)
       {
           //printf("Program Executed as Expected\n");
           //printf("Time Taken by the program for %d Threads=%f\n",Threads,time);
           printf("%f\n",time);
       }
       else
       {
           printf("Vector Addition hasnt been done properly,Mismatch in Values!!!\n");
       }
      
 
}
 
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
