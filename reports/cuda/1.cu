
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
 
#define n_size 100000
 
__global__ void add(float *a, float *b, float *c) {
 int index=threadIdx.x+blockIdx.x*blockDim.x;
 c[index]=a[index]+b[index];
}
 
void random_init(float a[],int ch)
{
   srand(time(NULL));
   if(ch==0)
   {
       for(int i=0;i<n_size;i++)
       {
           a[i]=((float)rand()/(float)(RAND_MAX)) * 5.0;
       }
   }
   else
   {
       for(int i=0;i<n_size;i++)
       {
             a[i]=(i+1);
       }
   }
  
}
 
int main() {
  
float a[n_size], b[n_size],c[n_size];
hipEvent_t start, end;
 
// host copies of variables a, b & c
float *d_a, *d_b, *d_c;
 
// device copies of variables a, b & c
int size = n_size*sizeof(float);
 
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
 
// Create Event for time
hipEventCreate(&start);
hipEventCreate(&end);
 
 
// Setup input values 
random_init(a,0);
random_init(b,0);
 
 
 
// Copy inputs to device
hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
int Thread[]={1,2,4,6,8,10,12,16,20,32,64,128,150};
int thread_arr_size=13;
 
for(int i=0;i<thread_arr_size;i++)
{
       int Threads=Thread[i];
       hipEventRecord(start);
 
       // Launch add() kernel on GPU
       add<<<n_size/Threads,Threads>>>(d_a, d_b, d_c);
 
       hipEventRecord(end);
       hipEventSynchronize(end);
 
       float time = 0;
       hipEventElapsedTime(&time, start, end);
 
       // Copy result back to host
       hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
       if(err!=hipSuccess) {
           printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
       }
           int flag=0;
       for(int i=0;i<n_size;i++)
       {
           if(c[i]!=(a[i]+b[i]))
           { 
               flag=1;
           break;
           }
       }
       if(flag==0)
       {
           printf("Program Executed as Expected\n");
           printf("Time Taken by the program for %d Threads=%f\n",Threads,time);
       }
       else
       {
           printf("Vector Addition hasnt been done properly,Mismatch in Values!!!\n");
       }
      
 
}
 
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
 
